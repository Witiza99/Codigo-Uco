
#include <hip/hip_runtime.h>

__global__ void my_kernel(float * z, float * y, float * x, int alpha, int vector_len){
	//Rutina Saxpy
	for(int i=0; i < vector_len; i++){
		z[i] = alpha * x[i] + y[i];
	}
}
