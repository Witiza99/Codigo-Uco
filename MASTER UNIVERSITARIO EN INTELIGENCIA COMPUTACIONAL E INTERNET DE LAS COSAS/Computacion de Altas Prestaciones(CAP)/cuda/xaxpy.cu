//Cabeceras útiles
#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>
#include <hip/hip_runtime.h>

//#include <cuda_utils.h>

/*
all: saxpy

saxpy: saxpy_cuda.o saxpy.o
        g++ -O3 -fopenmp saxpy.o saxpy_cuda.o -o driver_saxPy -I ./ -lm -lcudart -L/usr/local/cuda/lib64/

saxpy_cuda.o:
        nvcc -O3 -c saxpy_cuda.cu 
saxpy.o:
        g++ -O3 -fopenmp -c saxpy.c -lm -I./
clean:
        rm *.o
        rm driver_saxPy
*/

__global__ void my_kernel(float * z, float * y, float * x, int alpha, int vector_len){
	//Rutina Saxpy
	for(int i=0; i < vector_len; i++){
		z[i] = alpha * x[i] + y[i];
	}
}

//Rutina temporización (tiempo en segundos)
double dclock() {
	struct timeval tv;
	gettimeofday( &tv, NULL );
	return (double) (tv.tv_sec + tv.tv_usec*1.0e-6);
}

//Programa principal
int main() {
	const size_t vector_len = (32 * 1024 * 1024) / sizeof(float);	

	//Declaración y reserva de memoria (z, y, x)
	float alpha = 2;	
	float *z, *x, *y, *cuda_z, *cuda_x, *cuda_y;
	z = (float *)malloc(vector_len * sizeof(float));
	x = (float *)malloc(vector_len * sizeof(float));
	y = (float *)malloc(vector_len * sizeof(float));
	
	//reserva de memoria en gpu
	hipMalloc((void**)&cuda_z,(vector_len * sizeof(float)));
	hipMalloc((void**)&cuda_x,(vector_len * sizeof(float)));
	hipMalloc((void**)&cuda_y,(vector_len * sizeof(float)));
	if(z ==0 || y == 0 || x ==0 || cuda_z == 0 || cuda_x == 0 || cuda_y == 0)
		printf("Error con la reserva de memoria");

	//inicializamos a 0
	hipMemset(cuda_z, 0, (vector_len * sizeof(float)));
	hipMemset(cuda_x, 0, (vector_len * sizeof(float)));
	hipMemset(cuda_y, 0, (vector_len * sizeof(float)));


	//inicialización de valores por defecto en x e y
	for(int i=0; i < vector_len; i++){
		x[i] = 1;
		y[i] = 1;

	}

	double t0 = dclock();

	//copia de memoria cpu a gpu
	hipMemcpy(z, cuda_z, (vector_len * sizeof(float)), hipMemcpyHostToDevice);
	hipMemcpy(x, cuda_x, (vector_len * sizeof(float)), hipMemcpyHostToDevice);
	hipMemcpy(y, cuda_y, (vector_len * sizeof(float)), hipMemcpyHostToDevice);

	//llamada al kernel
	int blockSize = 1;
	int gridSize = 1;
	
	my_kernel<<<gridSize, blockSize>>>(cuda_z, cuda_y, cuda_x, alpha, vector_len);
	
	//copia de gpu a cpu
	hipMemcpy(cuda_z, z, (vector_len * sizeof(float)), hipMemcpyDeviceToHost);
	hipMemcpy(cuda_x, x, (vector_len * sizeof(float)), hipMemcpyDeviceToHost);
	hipMemcpy(cuda_y, y, (vector_len * sizeof(float)), hipMemcpyDeviceToHost);
	
	//se libera memoria
	free(z);
	free(y);
	free(x);
	hipFree(cuda_z);
	hipFree(cuda_y);
	hipFree(cuda_x);

	double t1 = dclock();
	double tsaxpy = t1 - t0;
	printf("Tiempo rutina saxpy: %.4f\n", tsaxpy);

	double gflops = (vector_len * 2) / (1.0e+9 * tsaxpy);
	
	printf("Gflops-> %.4f\n", gflops);
	
	

	return(0);
}

